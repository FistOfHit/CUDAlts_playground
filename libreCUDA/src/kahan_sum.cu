#include "hip/hip_runtime.h"
#include <librecuda.h>
#include <cstring>
#include <iostream>
#include <fstream>
#include <vector>

__global__ void kahan_sum_kernel(const float* data, int size, float* result) {
    float sum = 0.0f;
    float c = 0.0f;  // A running compensation for lost low-order bits

    for (int i = 0; i < size; ++i) {
        float y = data[i] - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    *result = sum;
}

float kahan_sum(const float* data, int size) {
    // Allocate device memory (accessible by CPU)
    void *d_data, *d_result;
    libreCuMemAlloc(&d_data, size * sizeof(float), true);
    libreCuMemAlloc(&d_result, sizeof(float), true);

    // Copy data to device (now we can directly access it)
    std::memcpy(d_data, data, size * sizeof(float));

    // Load the CUDA module from a file
    LibreCUmodule module;
    std::ifstream input("kahan_sum_kernel.cubin", std::ios::binary);
    std::vector<uint8_t> buffer(std::istreambuf_iterator<char>(input), {});
    libreCuModuleLoadData(&module, buffer.data(), buffer.size());

    // Get the kernel function
    LibreCUFunction kernel;
    libreCuModuleGetFunction(&kernel, module, "kahan_sum_kernel");

    // Set up kernel parameters
    void* params[] = {&d_data, &size, &d_result};

    // Create a stream
    LibreCUstream stream;
    libreCuStreamCreate(&stream, 0);

    // Launch the kernel
    libreCuLaunchKernel(kernel, 1, 1, 1, 1, 1, 1, 0, stream, params, sizeof(params) / sizeof(void*), nullptr);

    // Wait for the kernel to complete
    libreCuStreamCommence(stream);
    libreCuStreamAwait(stream);

    // Copy result back to host (now we can directly access it)
    float result = *(float*)d_result;

    // Clean up
    libreCuMemFree(d_data);
    libreCuMemFree(d_result);
    libreCuStreamDestroy(stream);
    libreCuModuleUnload(module);

    return result;
}
