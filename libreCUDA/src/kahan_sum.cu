#include "hip/hip_runtime.h"
#include <librecuda.h>
#include <cstring>

__global__ void kahan_sum_kernel(const float* data, int size, float* result) {
    float sum = 0.0f;
    float c = 0.0f;  // A running compensation for lost low-order bits

    for (int i = 0; i < size; ++i) {
        float y = data[i] - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    *result = sum;
}

extern "C" {
    float kahan_sum(const float* data, int size) {
        // Allocate device memory
        void *d_data, *d_result;
        libreCuMemAlloc(&d_data, size * sizeof(float), true);
        libreCuMemAlloc(&d_result, sizeof(float), true);

        // Copy data to device
        libreCuMemcpyHtoD(d_data, data, size * sizeof(float));

        // Load the CUDA module (assuming you have compiled the kernel to a cubin file)
        LibreCUmodule module;
        const char* module_name = "kahan_sum.cubin"; // Make sure this file exists
        libreCuModuleLoad(&module, module_name);

        // Get the kernel function
        LibreCUfunction kernel;
        libreCuModuleGetFunction(&kernel, module, "kahan_sum_kernel");

        // Set up kernel parameters
        void* params[] = {&d_data, &size, &d_result};

        // Create a stream
        LibreCUstream stream;
        libreCuStreamCreate(&stream, 0);

        // Launch the kernel
        libreCuLaunchKernel(kernel, 1, 1, 1, 1, 1, 1, 0, stream, params, sizeof(params) / sizeof(void*), nullptr);

        // Wait for the kernel to complete
        libreCuStreamCommence(stream);
        libreCuStreamAwait(stream);

        // Copy result back to host
        float result;
        libreCuMemcpyDtoH(&result, d_result, sizeof(float));

        // Clean up
        libreCuMemFree(d_data);
        libreCuMemFree(d_result);
        libreCuStreamDestroy(stream);
        libreCuModuleUnload(module);

        return result;
    }
}